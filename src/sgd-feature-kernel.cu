#include "hip/hip_runtime.h"
#include "sgd-feature-kernel.h"

__global__ void SgdFeature(const int* __restrict__ user_ids,
                           const int* __restrict__ item_ids,
                           const float* __restrict__ ratings,
                           const int num_features,
                           const float learning_rate,
                           const float regularization_factor,
                           float* __restrict__ user_features,
                           float* __restrict__ item_features) {
  // Compute error
  const int64_t user_id = user_ids[blockIdx.x];
  const int64_t item_id = item_ids[blockIdx.x];
  const int64_t user_feature_idx = user_id * num_features + threadIdx.x;
  const int64_t item_feature_idx = item_id * num_features + threadIdx.x;
  const float user_feature = user_features[user_feature_idx];
  const float item_feature = item_features[item_feature_idx];
  extern __shared__ float partial_sums[];
  partial_sums[threadIdx.x] = user_feature * item_feature;
  __syncthreads();
  if (threadIdx.x < warpSize) {
    for (int i = threadIdx.x + warpSize; i < blockDim.x; i += warpSize) {
      partial_sums[threadIdx.x] += partial_sums[i];
    }
  }
  __syncthreads();
  float predicted_rating = partial_sums[threadIdx.x];
  if (threadIdx.x < warpSize) {
    for (int i = warpSize / 2; i >= 1; i /= 2) {
      predicted_rating += __shfl_xor(predicted_rating, i, warpSize);
    }
  }
  __syncthreads();
  __shared__ float error;
  if (threadIdx.x == 0) {
    error = predicted_rating - ratings[blockIdx.x];
  }
  __syncthreads();
  // Update features
  const float new_user_feature = user_feature - learning_rate *
      (regularization_factor * user_feature + error * item_feature);
  const float new_item_feature = item_feature - learning_rate *
      (regularization_factor * item_feature + error * user_feature);
  user_features[user_feature_idx] = new_user_feature;
  item_features[item_feature_idx] = new_item_feature;
}

