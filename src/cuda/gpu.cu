#include "gpu.h"

// C++ headers
#include <cassert>

Gpu::Gpu(const int id, Logger* logger) : id_(id), logger_(logger) {
  hipError_t e = hipSuccess;
  int num_gpus = 0;
  e = hipGetDeviceCount(&num_gpus);
  logger_->CheckCudaError(e);
  assert(num_gpus > 0);
  e = hipSetDevice(id_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&shader_stream_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&h2d_stream_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&d2h_stream_);
  logger_->CheckCudaError(e);
}

Gpu::~Gpu() {
}

