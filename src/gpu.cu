#include "gpu.h"

Gpu::Gpu(const int id, Logger* logger) : id_(id), logger_(logger) {
  hipError_t e = hipSuccess;
  e = hipStreamCreate(&shader_stream_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&h2d_stream_);
  logger_->CheckCudaError(e);
  e = hipStreamCreate(&d2h_stream_);
  logger_->CheckCudaError(e);
}

Gpu::~Gpu() {
}

